#include "hip/hip_runtime.h"
// 

#include "filter_cuda.h"

#include "config.h"

#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "nvidia_helpers/hip/hip_runtime_api.h"

#include "utils.h"
#include "benchmark.h"
#include <iostream>

__global__ void filter_cuda_hist_kernel(const uint8_t* dev_input,
                                              uint8_t* dev_output,
                                        const int width,
                                        const int height,
                                        const int edge_x,
                                        const int edge_y,
                                        const int windowRadius,
                                        const int windowLength)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x<edge_x || x >= width-edge_x || y<edge_y || y >= height-edge_y)
    {
        return;
    }

    short hist[256];

    //clear hist
    for (int i = 0; i < 256; i++)
    {
        hist[i] = 0;
    }

    //fill hist
    for (int iy = y - windowRadius; iy <= y + windowRadius; iy++)
    {
        for (int ix = x - windowRadius; ix <= x + windowRadius; ix++)
        {
            hist[dev_input[iy*width + ix]]++;
        }
    }

    int cpt = 0;
    uint8_t ic = 0;

    //parse hist
    for (ic = 0; ic < 256; ic++)
    {
        cpt += hist[ic];

        if (cpt > windowLength/2)
        {
            break;
        }
    }

    dev_output[y*width + x] = ic;
}

void filter_cuda_hist_channel(uint8_t* data, const int w, const int h, const int windowSize)
{
    const int size = w * h * sizeof(uint8_t);
    int windowRadius = windowSize / 2;

    //memory allocation
    uint8_t* device_input_data = nullptr;
    uint8_t* device_output_data = nullptr;

    checkCudaErrors(hipMalloc((void **)& device_input_data, size));
    checkCudaErrors(hipMalloc((void **)& device_output_data, size));
    checkCudaErrors(hipMemcpy(device_input_data, data, size, hipMemcpyHostToDevice));

    dim3 dimBlock(CUDA_TILE_SIZE_X, CUDA_TILE_SIZE_Y,1);
    dim3 dimGrid(w / CUDA_TILE_SIZE_X +1 , h / CUDA_TILE_SIZE_Y +1,1);

    filter_cuda_hist_kernel << <dimGrid, dimBlock >> >(device_input_data, device_output_data, w, h, windowSize/2,windowSize/2, windowRadius,windowSize*windowSize);
   
    checkCudaErrors(hipPeekAtLastError());

    checkCudaErrors(hipMemcpy(data, device_output_data, size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(device_input_data));
    checkCudaErrors(hipFree(device_output_data));
  
}

void filter_cuda_hist(ImageData* img, unsigned int windowSize)
{


    ImageData* img_extended = image_extend(img,windowSize);

    if (img->isColor)
    {
        for (unsigned int i = 0; i < ImageData::CHANNEL_COUNT_RGB; i++)
        {          
            filter_cuda_hist_channel(img_extended->data_uint8[i],img_extended->width, img_extended->height, windowSize);
        }
    }
    else
    {
        filter_cuda_hist_channel(img_extended->data_uint8[3], img_extended->width, img_extended->height, windowSize);
    }

    image_placeExtendedBack(img,img_extended,windowSize);

    delete img_extended;
}