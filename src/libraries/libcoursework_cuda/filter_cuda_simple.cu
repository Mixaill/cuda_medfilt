#include "hip/hip_runtime.h"
// JackOLantern, PD
// Joseph Loser & Duncan Hill, MIT

#include "filter_cuda.h"

#include "config.h"

#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "nvidia_helpers/hip/hip_runtime_api.h"

//Kernels
__global__
void filter_cuda_simple_kernel(const uint8_t * data_dev_input, uint8_t * data_dev_output, const unsigned int width, const unsigned int height, const unsigned int windowSize) {
    //window radius & length
    const int windowRadius = windowSize / 2;
    const int windowLength = windowSize * windowSize;
        
    //current coordinates
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x > (width - 1) || y > (height - 1))
    {
        return;
    }

    uint8_t* device_window = new uint8_t[windowLength];

    // Populate the filter_array
    int filter_array_index = 0;
    for (int iy = y - windowRadius; iy <= y + windowRadius; iy++)
    {
        if (iy<0 || iy>(height - 1))
        {
            continue;
        }

        for (int ix = x - windowRadius; ix <= x + windowRadius; ix++)
        {
            if (ix<0 || ix>(width - 1))
            {
                continue;
            }

            device_window[filter_array_index++] = data_dev_input[iy*width + ix];
        }
    }

    //sort filter array
    uint8_t tmp;
    for (int i = 0; i <= windowLength / 2; i++)
    {
        for (int j = i + 1; j < windowLength; j++)
        {
            if (device_window[i] > device_window[j])
            { 
                tmp = device_window[i];
                device_window[i] = device_window[j];
                device_window[j] = tmp;
            }
        }
    }

    data_dev_output[y*width + x] = device_window[windowLength / 2];

    delete[] device_window;
}

void filter_cuda_simple_channel(uint8_t* data, const int w, const int h, const int windowSize)
{
    const int size = w * h * sizeof(uint8_t);

    //memory allocation
    uint8_t* device_input_data = nullptr;
    uint8_t* device_output_data = nullptr;

    checkCudaErrors(hipMalloc((void **)& device_input_data, size));
    checkCudaErrors(hipMalloc((void **)& device_output_data, size));
    checkCudaErrors(hipMemcpy(device_input_data, data, size, hipMemcpyHostToDevice));

    dim3 dimBlock(CUDA_TILE_SIZE_X, CUDA_TILE_SIZE_Y, 1);
    dim3 dimGrid(w / CUDA_TILE_SIZE_X+1, h / CUDA_TILE_SIZE_Y+1, 1);

    filter_cuda_simple_kernel << <dimGrid, dimBlock >> >(device_input_data, device_output_data, w, h, windowSize);

    checkCudaErrors(hipPeekAtLastError());

    checkCudaErrors(hipMemcpy(data, device_output_data, size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(device_input_data));
    checkCudaErrors(hipFree(device_output_data));

}

void filter_cuda_simple(ImageData* img, unsigned int windowSize) {

    if (img->isColor)
    {
        for (unsigned int i = 0; i < ImageData::CHANNEL_COUNT_RGB; i++)
        {
            filter_cuda_simple_channel(img->data_uint8[i], img->width, img->height, windowSize);
        }
    }
    else
    {
        filter_cuda_simple_channel(img->data_uint8[3], img->width, img->height, windowSize);
    }
}