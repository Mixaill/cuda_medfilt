#include "cudahelper.h"

#include <hip/hip_runtime.h>

#include "nvidia_helpers/hip/hip_runtime_api.h"
#include "nvidia_helpers/helper_string.h"

#include <iostream>

//Initialize CUDA device with highest Gflops value
int cw_cuda_DeviceInit()
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        std::cerr << "CUDA error: no devices supporting CUDA." << std::endl;
        exit(EXIT_FAILURE);
    }

    int dev = gpuGetMaxGflopsDeviceId();
    hipDeviceProp_t deviceProp;

    checkCudaErrors(hipSetDevice(dev));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", dev, deviceProp.name, deviceProp.major, deviceProp.minor);

    return dev;
}

bool cw_cuda_printVersionInfo()
{
    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    // Min spec is SM 1.0 devices
    bool bVal = checkCudaCapabilities(1, 0);
    return bVal;
}